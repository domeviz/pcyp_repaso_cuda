#include <iostream>
#include <hip/hip_runtime.h>

#define VECTOR_ELEMENTS 2048

extern "C"
{
__declspec(dllexport) unsigned long NvOptimusEnablement = 0x00000001;
}

__global__
void vecAdd(float *d_A, float *d_B, float *d_C, int n) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;

    if (index < n)
        d_C[index] = d_A[index] + d_B[index];
}

int main() {

    // host
    float *h_A = new float[VECTOR_ELEMENTS];
    float *h_B = new float[VECTOR_ELEMENTS];
    float *h_C = new float[VECTOR_ELEMENTS];

    // Device
    float *d_A, *d_B, *d_C;
    int size = VECTOR_ELEMENTS * sizeof(float);
    hipMalloc(&d_A, size);
    hipMalloc(&d_B, size);
    hipMalloc(&d_C, size);

    // Inicializar vectores
    for (int i = 0; i < VECTOR_ELEMENTS; i++) {
        h_A[i] = 1.f;
        h_B[i] = 2.f;
        h_C[i] = 0.f;
    }

    // Linea para probar el consumo de memoria de la GPU
    //cudaMalloc(&xx, 1024 * 1024 * 1024 * sizeof(float));

    // Copiar host-to-device
    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    // invocar al kernel
    vecAdd<<<8, 256>>>(d_A, d_B, d_C, VECTOR_ELEMENTS);

    // copiar devide-to-host
    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    for (int i = 0; i < 15; i++) {
        printf("%.0f, ", h_C[i]);
    }

    return 0;
}